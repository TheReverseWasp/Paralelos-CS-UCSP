#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <string>
#include <vector>
#include <fstream>

#include "model.h"
//#include "modelh.h"
#include "invoker.h"

#define uint unsigned int

using namespace std;

__device__ __host__
float add (pair <string, int> to_add, modelh my_model) {
  my_model.head -> add (to_add);
  return 1.0;
}

__device__ __host__
float search(string to_search, modelh my_model) {
  my_model.head -> search (to_search);
  return 1.0;
}

__global__
void controller(modelh *my_model, vector <pair <string, int> > *to_run) {
  uint tpos = blockIdx.x*blockDim.x+threadIdx.x;
  for (uint i = 0; i < to_run[tpos].size(); i++) {
    add (to_run[tpos][i], my_model[tpos]);
  }
}

__global__
void searcher1 (modelh *my_model, string to_search, uint arr_pos, uint &answer) { ///vagos
  uint tpos = blockIdx.x*blockDim.x+threadIdx.x;
  if (arr_pos == tpos) {
    if (search (to_search, my_model[arr_pos]) > 0) {
      answer = arr_pos;
    }
  }
}

__global__
void searcher2 (modelh *my_model, string to_search, uint &answer) { ///all run
  uint tpos = blockIdx.x*blockDim.x+threadIdx.x;
  if (search (to_search, my_model[tpos]) > 0) {
    answer = tpos;
  }
}

__global__
void searcher3  (modelh *my_model, string to_search, uint arr_pos, uint &answer) { ///one run
  if (search (to_search, my_model[arr_pos]) > 0) {
    answer = arr_pos;
  }
}

void tempdoll() {
  modelh *my_model = new modelh[256];
  int size = 256 * sizeof(modelh);
  vector<pair <string, int> >word_l[256];
  size = 256 * sizeof(vector<pair <string, int> >);
  ifstream my_file("dataset.txt");
  int c = 0;
  string line;
  pair <string, int> runner;
  while (getline (my_file, line)) {
    runner.first = line;
    runner.second = c;
    word_l[c % 256].push_back(runner);
    ++c;
  }
  my_file.close();
  //hipMalloc((void**) &word_l, size);
  ///
  for (uint i = 0; i < 256; i++) {
    for (uint j = 0; j < word_l[i].size(); j++) {
      my_model -> head -> add (word_l[i][j]);
    }
  }

  ///

  hipMalloc((void**) &my_model, size);
  ///
  dim3 dimGrid(256);
  dim3 dimBlock(1);
  //controller <<< dimGrid, dimBlock >>> (my_model, word_l);
  ///
  uint answer = 500;
  searcher1 <<< dimGrid, dimBlock >>> (my_model, "which", 0, answer);
  if (answer != 500) {
    cout << "encontrado en " << answer <<endl;
  }
  answer = 500;
  searcher2 <<< dimGrid, dimBlock >>> (my_model, "which", answer);
  if (answer != 500) {
    cout << "encontrado en " << answer <<endl;
  }
  answer = 500;
  searcher3 <<< 1, 1 >>> (my_model, "which", 0, answer);
  if (answer != 500) {
    cout << "encontrado en " << answer <<endl;
  }

  ///
  hipFree(my_model); ///hipFree(word_l);
}

int main(int argc, char const *argv[]) {
  tempdoll();
  return 0;
}
