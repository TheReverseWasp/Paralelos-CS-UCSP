#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

////1.2
__global__
void multmatrix(float *A, float *B, float *C, int maxlength){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i < matlength && j < matlength) {
    C[i*matlength+j]=A[i*matlength+j]+B[i*matlength+j];
    ///1.3
    /*
    before if
    int j;
    for (j = 0; j < maxlength; j++) {
      C[i*matlength+j] = A[i*matlength+j] + B[i*matlength+j];
    }
    */
    ///1.4
    /*
    before if
    int i;
    for (i = 0; i < maxlength; i++) {
      C[i*matlength+j] = A[i*matlength+j] + B[i*matlength+j];
    }
    */

  }
}
/*
dim3 threads(16,16);
dim3 blocks((maxlength+15)/16, (maxlength+15)/16);

*/

////1.1
void multmatrix(float *h_A, float *h_B, float *h_C, int matlength) {
  float *d_A, *d_B, *d_C;
  int size = maxlength * maxlength *sizeof(float);
  hipMalloc((void**)d_A,size);
  hipMalloc((void**)d_B,size);
  hipMalloc((void**)d_C,size);

  hipMemcpy(d_A,h_A,size, hipMemcpyHostToDevice);
  hipMemcpy(d_B,h_B,size, hipMemcpyHostToDevice);
  hipMemset(d_C, 0, size);

  dim3 threads(__,__,__);
  dim3 blocks(__,__,__);
  mulmatrixkernel<<<blocks,threads>>>(d_A,d_B,d_C,matlength);

  hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
  hipFree(d_A);hipFree(d_B);hipFree(d_C);
}
