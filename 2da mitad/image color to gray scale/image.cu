#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/imgproc/imgproc.hpp>
using namespace std;

int BLUR_SIZE = 5;
int height = 800;
int width = 800;

__global__
void blurKerner(unsigned char * in, unsigned char * out,
int w, int h)
{
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  ////
  if (Col < w && Row < h) {
    int pixVal = 0;
    int pixels = 0;
    for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1
      ; blurRow++) {
      for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1;
         blurCol++) {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        if (curRow > -1 && curRow < h && curCol > -1 &&
        curCol < w) {
          pixVal +=in[curRow * w + curCol];
          pixels++;
        }
      }
    }
    out[Row * w + Col] = unsigned char(pixVal / pixels);
  }
}
////
void tempdoll(unsigned char *buffer,
  unsigned char *blured_ret, int w, int h){
  int size = w * h * sizeof(unsigned char);
  unsigned char blured[w * h], grayed[w * h],
  D_b[w * h];
  ///
  hipMalloc((void**) &D_b, size);
  hipMemcpy(D_b, buffer, hipMemcpyHostToDevice);
  hipMalloc((void**) &blured, size);
  ///
  blurKerner <<< ceil((w * h)/256.0), 256 >>> (D_b, blured, w, h);
  hipMemcpy(blured_ret, blured, hipMemcpyDeviceToHost);
  ///
  hipFree(D_b); hipFree(blured); //hipFree(D_b);
}

//// convertir a Mat un arreglo
Mat convertToMat(unsigned char *buffer) {
  Mat tmp(width, height, CV_8UC1);
  for (int x = 0; x < height; x++) {
    for (int y = 0; y < width; y++) {
      int value = (int) buffer[x * width + y];
      tmp.at<int>(y, x) = value;
    }
  }
  return tmp;
}







int main(int argc, char const *argv[]) {
  unsigned char *Pout
  char* imageName = argv[1];
  Mat image;
  image = imread( imageName, 1 );
  ///

  if( argc != 2 || !image.data )
  {
   printf( " No image data \n " );
   return -1;
  }
  ///

  unsigned char buffer[height * width];
  unsigned char out_blur[height * width];
  unsigned char out_gray[height * width];
  for (int j = 0; j < height; j++) {
    for (int i = 0; i < width; i++) {
      uchar& uxy = image.at<uchar>(j, i);
      unsigned char color = (unsigned char) uxy;
      buffer[j * width + i] = color;
    }
  }
  ///
  tempdoll(buffer, out_blur, width, height);

  Mat blured_img = convertToMat(out_blur);
  imwrite( "blured_Image.jpg", blured_img );
  return 0;
}
