#include "hip/hip_runtime.h"
#include <stdio.h>


__global__
void colorToGreyscaleConversion(unsigned char *Pout, unsigned,
  char *Pin, int width, int height) {
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    if (Col < width && Row < height) {
      // get 1D coordinate for the grayscale image
      int greyOffset = Row*width + Col;
      // one can think of the RGB image having
      // CHANNEL times columns than the grayscale image
      int rgbOffset = greyOffset*CHANNELS;
      unsigned char r = Pin[rgbOffset]; // red value for pixel
      unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
      unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel
      // perform the rescaling and store it
      // We multiply by floating point constants
      Pout[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}
