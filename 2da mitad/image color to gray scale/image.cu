#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>

#include "lodepng.h"
#include "helperfuns.h"
using namespace std;

__global__
void blurKerner(unsigned char * in, unsigned char * out,
int w, int h)
{
  int BLUR_SIZE = 5;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  ////
  if (Col < w && Row < h) {
    int pixVal = 0;
    int pixels = 0;
    for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1
      ; blurRow++) {
      for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1;
         blurCol++) {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        if (curRow > -1 && curRow < h  && curCol > -1 &&
        curCol < w) {
          pixVal +=in[curRow * w + curCol];
          pixels++;
        }
      }
    }
    out[Row * w + Col] = (unsigned char)(pixVal / pixels);
  }
}
////grayscale function
__global__
void colorToGreyscaleConversion(unsigned char * Pout, unsigned
  char * Pin, int width, int height) {
  int Col = threadIdx.x + blockIdx.x * blockDim.x;
  int Row = threadIdx.y + blockIdx.y * blockDim.y;
  if (Col < width && Row < height) {
  // get 1D coordinate for the grayscale image
    int greyOffset = Row*width + Col;
    // one can think of the RGB image having
    // CHANNEL times columns than the grayscale image
    int rgbOffset = greyOffset * 3;
    unsigned char r = Pin[rgbOffset  ]; // red value for pixel
    unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
    unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel
    // perform the rescaling and store it
    // We multiply by floating point constants
    Pout[greyOffset] = 0.21f*r + 0.07f*b + 0.71f*g;
  }
}

////temper function
void tempdoll(unsigned char *buffer,
  unsigned char *blured_ret, unsigned char *grayed_ret, int w, int h){
  int size = w * h * sizeof(unsigned char);
  unsigned char *blured = new unsigned char[w * h],
  *grayed = new unsigned char[w * h], *D_b = new unsigned char[w * h];
  ///
  hipMalloc((void**) &D_b, size);
  hipMemcpy(D_b, buffer, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &blured, size);
  hipMalloc((void**) &grayed, size);
  ///
  dim3 dimBlock(256, 1, 1);
  dim3 dimBlock(16, 16, 1);
  blurKerner <<< ceil((w * h)/256.0), dimBlock >>> (D_b, blured, w, h);
  colorToGreyscaleConversion <<<
   ceil((w * h) / 256.0), dimBlock >>>
   (grayed, D_b, w, h);
  hipMemcpy(blured_ret, blured, size, hipMemcpyDeviceToHost);
  hipMemcpy(grayed_ret, grayed, size, hipMemcpyDeviceToHost);
  ///
  hipFree(D_b); hipFree(blured); hipFree(grayed);
}

int main(int argc, char const *argv[]) {
  vector <unsigned char> buffer;
  unsigned w1,h1;
  const char* filename = "cat.png";
  //decode
  unsigned error = lodepng::decode(buffer, w1, h1, filename);
  int w = w1, h = h1;
  //if there's an error, display it
  if(error) std::cout << "decoder error " << error << ": " << lodepng_error_text(error) << std::endl;
  //the pixels are now in the vector "image", 4 bytes per pixel, ordered RGBARGBA..., use it as texture, draw it, ...
  vector <unsigned char> blured(buffer.size()), gray(buffer.size());
  unsigned char *buffer1 = new unsigned char[w*h*4], *blured1 = new unsigned char[w*h*4],
  *gray1 = new unsigned char[w*h*4];
  vectoarr(buffer, buffer1);
  ///
  tempdoll(buffer1, blured1, gray1, w, h);
  ////
  arrtovec(blured1, blured); arrtovec(gray1, gray);
  ///

  vector<unsigned char> png1, png2;
  error = lodepng::encode(png1, blured, w, h);
  if(!error) lodepng::save_file(png1, "catb.png");
  //if there's an error, display it
  if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;


  error = lodepng::encode(png2, gray, w, h);
  if(!error) lodepng::save_file(png2, "catg.png");
  //if there's an error, display it
  if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;




  return 0;
}
