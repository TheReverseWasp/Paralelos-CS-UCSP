
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if ((Row < Width) && (Col < Width)) {
        float Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
        P[Row*Width+Col] = Pvalue;
    }
}





int main()
{
    const int w = 2;
    float M[w][w] = {{0, 1}, {2, 3}};
    float N[w][w] = {{0, 1}, {2, 3}};
    float P[w][w];

    int sizeMat = w * w * sizeof(float);
    float *d_M, *d_N, *d_P;

    hipMalloc((void **) &d_M, w * w * sizeof(float));
    hipMalloc((void **) &d_N, w * w * sizeof(float));
    hipMalloc((void **) &d_P, w * w * sizeof(float));

    hipMemcpy(d_M, M, sizeMat, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, sizeMat, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, sizeMat, hipMemcpyHostToDevice);

    dim3 dimBlock(w, w, 1);
    MatrixMulKernel<<<1, dimBlock>>>(d_M, d_N, d_P, w);

    hipMemcpy(P, d_P, sizeMat, hipMemcpyDeviceToHost);

    for (int i = 0; i < w; ++i)
    {
        for (int j = 0; j < w; ++j)
        {
            std::cout << P[i][j] << " ";
        }
        std::cout << "\n";
    }

    return 0;
}
