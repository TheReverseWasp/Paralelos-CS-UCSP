#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 2

__global__ void MatrixMulTileKernel(float* d_M, float* d_N, float* d_P, int Width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

     // Identify the row and column of the d_P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    // Loop over the d_M and d_N tiles required to compute d_P element
    for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {
     // Collaborative loading of d_M and d_N tiles into shared memory
        Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
        Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();

    }
    d_P[Row*Width + Col] = Pvalue;
}



int main() {
    const int w = 4;
    float M[w][w] = {{1,0,0,0},{0,1,0,0},{0,0,1,0},{0,0,0,1}};
    float N[w][w] = {{0,1,2,3},{4,5,6,7},{8,9,10,11},{12,13,14,15}};
    float P[w][w];

    int sizeMat = w * w * sizeof(float);
    float *d_M, *d_N, *d_P;

    hipMalloc((void **) &d_M, w * w * sizeof(float));
    hipMalloc((void **) &d_N, w * w * sizeof(float));
    hipMalloc((void **) &d_P, w * w * sizeof(float));

    hipMemcpy(d_M, M, sizeMat, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N, sizeMat, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, sizeMat, hipMemcpyHostToDevice);

    dim3 dimGrid(2, 2, 1);
    dim3 dimBlock(2, 2, 1);
    MatrixMulTileKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, w);

    hipMemcpy(P, d_P, sizeMat, hipMemcpyDeviceToHost);

    for (int i = 0; i < w; ++i) {
        for (int j = 0; j < w; ++j) {
            std::cout << P[i][j] << " ";
        }
        std::cout << "\n";
    }

    return 0;
}
