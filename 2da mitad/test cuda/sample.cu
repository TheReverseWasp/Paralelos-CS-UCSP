
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  if(i < n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;
  ///
  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_C, size);

  vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  ///Free device memory for A, B, C
  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
int main(int argc, char const *argv[]) {
  float *A, *B, *C;
  int n = 10000;
  A = new float[n];
  B = new float[n];
  C = new float[n];
  for (int i = 0; i < n; i++) {
    A[i] = i / 100.0;
    B[i] = i;
    C[i] = 0;
  }
  vecAdd (A, B, C, n);
  for (int i = 0; i < n; i++) {
    printf("%f\n", C[i]);
  }
  return 0;
}
