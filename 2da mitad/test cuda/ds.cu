#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <iostream>
#include <utility>
#include <string>

using namespace std;

int *c_p = new int[3];
c_p[0] = c_p[1] = c_p[2] = 0;

__global__
void binary_search (pair <string, int> *C, pair <string, int> *B,
  pair <string, int> *C, int n, string elem) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int half = n / 2;
  int fin = n, ini = 0;
  if (i < 3) {
    if (i == 0) {
      while (ini - fin > 1) {
        if (A[half].first == elem) {
          cout << "encontrada concurrencia en A: " << A[half].second << endl;
          return;
        }
        else if (A[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (A[ini].first == elem) {
        cout << "encontrada concurrencia en A: " << A[ini].second << endl;
        return;
      }
    }
    else if (i == 1) {
      half *= 2;
      fin *= 2;
      while (ini - fin > 1) {
        if (B[half].first == elem) {
          cout << "encontrada concurrencia en B: " << B[half].second << endl;
          return;
        }
        else if (B[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (B[ini].first == elem) {
        cout << "encontrada concurrencia en B: " << B[ini].second << endl;
        return;
      }
    }
    else {
      half *= 4;
      fin *= 4;
      while (ini - fin > 1) {
        if (C[half].first == elem) {
          cout << "encontrada concurrencia en C: " << C[half].second << endl;
          return;
        }
        else if (C[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (C[ini].first == elem) {
        cout << "encontrada concurrencia en C: " << C[ini].second << endl;
        return;
      }
    }
  }
}

__global__
void update_LSM (pair <string, int> *C, pair <string, int> *B,
  pair <string, int> *C, int n, string elem, int new_val) {
  int i = blockDim.x*blockIdx.x + threadIdx.x;
  int half = n / 2;
  int fin = n, ini = 0;
  if (i < 3) {
    if (i == 0) {
      while (ini - fin > 1) {
        if (A[half].first == elem) {
          cout << "encontrada concurrencia en A: " << A[half].second << endl;
          A[half].second = new_val;
          return;
        }
        else if (A[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (A[ini].first == elem) {
        cout << "encontrada concurrencia en A: " << A[ini].second << endl;
        A[half].second = new_val;
        return;
      }
    }
    else if (i == 1) {
      half *= 2;
      fin *= 2;
      while (ini - fin > 1) {
        if (B[half].first == elem) {
          cout << "encontrada concurrencia en B: " << B[half].second << endl;
          B[half].second = new_val;
          return;
        }
        else if (B[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (B[ini].first == elem) {
        cout << "encontrada concurrencia en B: " << B[ini].second << endl;
        B[half].second = new_val;
        return;
      }
    }
    else {
      half *= 4;
      fin *= 4;
      while (ini - fin > 1) {
        if (C[half].first == elem) {
          cout << "encontrada concurrencia en C: " << C[half].second << endl;
          C[half].second = new_val;
          return;
        }
        else if (C[half].first < elem) {
          fin = half;
        }
        else {
          ini = half;
        }
        half = (ini + fin) / 2;
      }
      if (C[ini].first == elem) {
        cout << "encontrada concurrencia en C: " << C[ini].second << endl;
        C[half].second = new_val;
        return;
      }
    }
  }
}

void LSMInit(pair <string, int> *A, pair <string, int> *B, pair <string, int> *C, int n) {
  int size = n * sizeof(pair <string, int>);
  pair <string, int> *d_A, *d_B, *d_C;
  ///
  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);
  hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  ///vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);
  ///Free device memory for A, B, C
}

void LSMEND() {
  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main(int argc, char const *argv[]) {
  pair <string, int> *A, *B, *C;
  int sizu;
  cin >> sizu;
  A = new pair <string, int> [sizu + 5];
  B = new pair <string, int> [sizu * 2 + 5];
  C = new pair <string, int> [sizu * 4 + 5];
  LSMInit(A, B, C, sizu);
  ///
  /*

  Interaction part

  */
  LSMEND();
}
