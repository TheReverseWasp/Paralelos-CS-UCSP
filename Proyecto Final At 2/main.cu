#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <vector>
#include <utility>
#include <cuda.cu>
#include <fstream>

#define uint unsigned int
#define Leaves 256
#define defsize 2
#define arrsize 64 /// if dataset grows increse 50
#define levels 5
using namespace std;


/****************************************add to struct*****************************************************/
__device__
float add_itself (pair <string, int> **c_dis, int **actual_pos, pair <string, int> elem, uint lvl) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  uint eval = (lvl + 1) + defsize;
  if (actual_pos[tpos][lvl] == eval) {
    if (lvl == levels - 1) {
      return 0f;
    }
    for (uint i = 0; i < eval; i++) {
      add_itself (c_dis, actual_pos, c_dis[tpos][i], lvl + 1);
    }
    actual_pos[tpos][lvl] = 0;
    add_itself(c_dis, actual_pos, elem, lvl);
  }
  return 1f;
}

__global__
void add(pair <string, int> **c_dis, int **actual_pos, pair <string, int> *lines, uint sizelines) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  if (tpos < Leaves) {
    for (uint i = tpos; i < sizelines; i += Leaves) {
      add_itself (c_dis, actual_pos, lines[i], 0);
    }
  }
}
/****************************************search on struct*****************************************************/

__global__
void search (pair <string, int> **c_dis, int **actual_pos, string to_search, uint &threadID, uint &elemeID) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  if (tpos < Leaves) {
    for (uint j = 0; j < levels; j++) {
      for (uint k = 0; k < actual_pos[tpos][j]; k++) {
        if (to_search == c_dis[tpos][k].first) {
          threadID = tpos;
          elemeID = c_dis[tpos][k].second;
          return;
        }
      }
    }
  }
}





void tempdoll () {
  /////////////////////////////struct///////////////////////////////////////
  pair <string, int> **c_dis = new pair <string, int> *[Leaves];
  for (uint i = 0; i < Leaves; ++i) {
    c_dis[i] = new pair <string, int> [arrsize];
  }
  int **actual_pos = new int* [Leaves];
  for (uint i = 0; i < Leaves; i++) {
    actual_pos[i] = new int [levels];
    for (uint j = 0; j < levels; j++) {
      actual_pos[i][j] = 0;
    }
  }
  int sizepair = Leaves * arrsize * sizeof (pair <string, int>), sizepos = Leaves * levels * sizeof (Int);
  hipMalloc((void**) c_dis, sizepair);
  hipMalloc((void**) actual_pos, sizepos);
  /////////////////////////////input///////////////////////////////////////
  vector <string, int> linesonv;
  pair <string, int> temp;
  int c = 0;
  while (getline (my_file, temp.first)) {
    temp.second = c;
    linesonv.push_back (temp);
    ++c;
  }
  pair <string, int> *lines = new pair <string, int> [linesonv.size ()];
  for (uint i = 0; i < linesonv.size (); i++) {
    lines[i] = linesonv[i];
  }
  int sizeinput = linesonv.size() * sizeof(pair <string, int>);
  hipMalloc((void **) &lines, sizeinput);
  /////////////////////////////add elements///////////////////////////////////////
  dim3 dimGrid(256);
  dim3 dimBlock(1);       ///modificar en caso de aumentar los threads a usar
  add <<< dimGrid, dimBlock >>> (c_dis, actual_pos, lines, c);
  /////////////////////////////search elements///////////////////////////////////////
  uint threadID, elemeID;
  string to_search = "gooks";
  search <<< dimGrid, dimBlock >>> (c_dis, actual_pos, to_search, threadID, elemeID);
  cout << to_search << "found on thread " << threadID << " and its value is " << elemeID <<endl;
}



/**/
int main(int argc, char const *argv[]) {
  tempdoll();
  return 0;
}
