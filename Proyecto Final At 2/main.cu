#include <iostream>
#include <algorithm>
#include <vector>
#include <utility>
#include <hip/hip_runtime.h>
#include <fstream>

#define uint unsigned int
#define Leaves 256
#define defsize 2
#define arrsize 64 /// if dataset grows increse 64
#define levels 5

using namespace std;


/****************************************add to struct*****************************************************/

__device__
void add_itself (pair <char*, int> **c_dis, int **actual_pos, pair <char*, int> elem, uint lvl) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  uint eval = (lvl + 1) * defsize;
  uint peval = lvl * defsize;
  if (actual_pos[tpos][lvl] == eval) {
    if (lvl == levels - 1) {
      return;
    }
    for (uint i = 0; i < eval; i++) {
      (add_itself (c_dis, actual_pos, c_dis[tpos][eval - peval + i], lvl + 1));
    }
    actual_pos[tpos][lvl] = 0;
    (add_itself (c_dis, actual_pos, elem, lvl));
  }
  return;
}

__global__
void add (pair <char*, int> **c_dis, int **actual_pos, pair <char*, int> *lines, uint sizelines) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  if (tpos < Leaves) {
    for (uint i = tpos; i < sizelines; i += Leaves) {
      add_itself (c_dis, actual_pos, lines[i], 0);
    }
  }
}
/****************************************search on struct*****************************************************/

__global__
void search (pair <char*, int> **c_dis, int **actual_pos, char* to_search, uint &threadID, uint &elemeID) {
  uint tpos = blockIdx.x * blockDim.x + threadIdx.x;
  if (tpos < Leaves) {
    for (uint j = 0; j < levels; j++) {
      for (uint k = 0; k < actual_pos[tpos][j]; k++) {
        if (to_search == c_dis[tpos][k].first) {
          threadID = tpos;
          elemeID = c_dis[tpos][k].second;
          return;
        }
      }
    }
  }
}





void tempdoll () {
  /////////////////////////////struct///////////////////////////////////////
  pair <char*, int> **c_dis = new pair <char*, int> *[Leaves];
  for (uint i = 0; i < Leaves; ++i) {
    c_dis[i] = new pair <char*, int> [arrsize];
  }
  int **actual_pos = new int* [Leaves];
  for (uint i = 0; i < Leaves; i++) {
    actual_pos[i] = new int [levels];
    for (uint j = 0; j < levels; j++) {
      actual_pos[i][j] = 0;
    }
  }
  int sizepair = Leaves * arrsize * sizeof (pair <char*, int>), sizepos = Leaves * levels * sizeof (int);
  hipMalloc((void**) c_dis, sizepair);
  hipMalloc((void**) actual_pos, sizepos);
  /////////////////////////////input///////////////////////////////////////
  vector <pair <char*, int> > linesonv;
  pair <char*, int> temp;
  temp.first = new char[6];
  string templine;
  int c = 0;
  ifstream my_file("dataset.txt");
  while (getline (my_file, templine)) {
    temp.second = c;
    strcpy(temp.first, templine.c_str());
    linesonv.push_back (temp);
    ++c;
  }
  my_file.close ();
  pair <char*, int> *lines = new pair <char*, int> [linesonv.size ()];
  for (uint i = 0; i < linesonv.size (); i++) {
    lines[i] = linesonv[i];
  }
  int sizeinput = linesonv.size() * sizeof(pair <char*, int>);
  hipMalloc((void **) &lines, sizeinput);
  /////////////////////////////add elements///////////////////////////////////////
  dim3 dimGrid(256);
  dim3 dimBlock(1);       ///modificar en caso de aumentar los threads a usar
  add <<< dimGrid, dimBlock >>> (c_dis, actual_pos, lines, c);
  /////////////////////////////search elements///////////////////////////////////////
  uint threadID, elemeID;
  char *to_search = "jukes";
  search <<< dimGrid, dimBlock >>> (c_dis, actual_pos, to_search, threadID, elemeID);
  cout << to_search << " found on thread " << threadID << " and its value is " << elemeID <<endl;
  ///////////////////////////////////////free///////////////////////////////////////
  hipFree (lines); hipFree (c_dis); hipFree(actual_pos);


}



/**/
int main(int argc, char const *argv[]) {
  tempdoll();
  return 0;
}
